// needs to be compiled with option -arch=sm_20 to work


#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_CHECK(cmd) {hipError_t error = cmd; if(error!=hipSuccess){printf("<%s>:%i ",__FILE__,__LINE__); printf("[CUDA] Error: %s\n", hipGetErrorString(error));}}
#define CUDA_CHECK_KERNEL {hipError_t error = hipGetLastError(); if(error!=hipSuccess){printf("<%s>:%i ",__FILE__,__LINE__); printf("[CUDA] Error: %s\n", hipGetErrorString(error));}}

#define N 1000000
#define DELTA 0.001f 

__global__ void init_c(float *c){
  int idx=blockIdx.x*blockDim.x+threadIdx.x;
  if (idx==0) *c=0.0f;
}

__global__ void scalarp(float *a, float *b, float *c){
  int idx=blockIdx.x*blockDim.x+threadIdx.x;
  float temp;

  if (idx<N) {
    temp=a[idx]*b[idx];
    atomicAdd(c,temp);
  }
}


int main (int argc, char **argv){

  float a_host[N], b_host[N], c_host, d_host=0.0f;
  float *a_device, *b_device, *c_device;
  int i;
  int blocksize=256;
  dim3 dimBlock(blocksize);
  dim3 dimGrid(ceil(N/(float)blocksize));

  for (i=0;i<N;i++) a_host[i]=1.0f*i;
  for (i=0;i<N;i++) b_host[i]=1.0f*i;

  CUDA_CHECK(hipMalloc((void**)&a_device,N*sizeof(float)));
  CUDA_CHECK(hipMalloc((void**)&b_device,N*sizeof(float)));
  CUDA_CHECK(hipMalloc((void**)&c_device,sizeof(float)));
  
  CUDA_CHECK(hipMemcpy(a_device,a_host,N*sizeof(float),hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(b_device,b_host,N*sizeof(float),hipMemcpyHostToDevice));

  init_c<<<1,1>>>(c_device);
  CUDA_CHECK_KERNEL

  scalarp<<<dimGrid,dimBlock>>>(a_device,b_device,c_device);
  CUDA_CHECK_KERNEL

  CUDA_CHECK(hipMemcpy(&c_host,c_device,sizeof(float),hipMemcpyDeviceToHost));

  for (i=0;i<N;i++) d_host+=a_host[i]*b_host[i];
  if ((abs(d_host - c_host) > DELTA*c_host)) printf("Solution invalid. GPU has %g, CPU has %g\n",c_host,d_host);

  CUDA_CHECK(hipFree(a_device));
  CUDA_CHECK(hipFree(b_device));
  CUDA_CHECK(hipFree(c_device));

  return 0;
}

