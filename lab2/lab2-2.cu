
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define N 1000000


__global__ void vecadd(int *a, int *b, int *c){
  // determine global thread id
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  // do vector add, check if index is < N
  if(idx<N) {
    c[idx]=a[idx]+b[idx];
  }
}

int main (int argc, char **argv){

  int a_host[N], b_host[N], c_host[N];
  int *a_device, *b_device, *c_device;
  int i;
  int blocksize=256;
  dim3 dimBlock(blocksize);
  dim3 dimGrid(ceil(N/(float)blocksize));

  for (i=0;i<N;i++) a_host[i]=i;
  for (i=0;i<N;i++) b_host[i]=i;

  // alloc GPU memory
  hipMalloc((void**)&a_device,N*sizeof(int));
  hipMalloc((void**)&b_device,N*sizeof(int));
  hipMalloc((void**)&c_device,N*sizeof(int));

  // transfer data
  hipMemcpy(a_device,a_host,N*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(b_device,b_host,N*sizeof(int),hipMemcpyHostToDevice);
    
    
  // invoke kernel
  vecadd<<<dimGrid,dimBlock>>>(a_device,b_device,c_device);

  // transfer result
  hipMemcpy(c_host,c_device,N*sizeof(int),hipMemcpyDeviceToHost);
 
  // check for correctness
  for (i=0;i<N;i++) assert (c_host[i] == a_host[i] + b_host[i]);

  // free GPU memory
  for (i=0;i<N;i++) {
    hipFree(a_device);
    hipFree(b_device);
    hipFree(c_device);
  } 
   
  return 0;
}

