
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define N 1000000

int main (int argc, char **argv){

  int a_host[N], b_host[N];
  int *a_device, *b_device;
  int i;
  
  // initialize data
  for (i=0;i<N;i++) {
    a_host[i]=i;
  }
  
  // allocate device memory
  hipMalloc((void**)&a_device,N*sizeof(int));
  hipMalloc((void**)&b_device,N*sizeof(int));
  
  // transfer data onto the device, copy on device, transfer back
  hipMemcpy(a_device,a_host,N*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(b_device,a_host,N*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(b_host,b_device,N*sizeof(int),hipMemcpyDeviceToHost);
 
  
  // correctness check
  for (i=0;i<N;i++) {
	assert (a_host[i]==b_host[i]);
  }
  // free GPU memory
  for (i=0;i<N;i++) {
    hipFree(a_device);
    hipFree(b_device);
  } 
  
  
  return 0;
}
