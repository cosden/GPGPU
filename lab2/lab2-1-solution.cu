
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define N 1000000

int main (int argc, char **argv){

  int a_host[N], b_host[N];
  int *a_device, *b_device;
  int i;

  for (i=0;i<N;i++) a_host[i]=i;

  hipMalloc((void**)&a_device,N*sizeof(int));
  hipMalloc((void**)&b_device,N*sizeof(int));

  hipMemcpy(a_device,a_host,N*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(b_device,a_device,N*sizeof(int),hipMemcpyDeviceToDevice);
  hipMemcpy(b_host,b_device,N*sizeof(int),hipMemcpyDeviceToHost);

  for (i=0;i<N;i++) assert (a_host[i]==b_host[i]);

  hipFree(a_device);
  hipFree(b_device);

  return 0;
}
